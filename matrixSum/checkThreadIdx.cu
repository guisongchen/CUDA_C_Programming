#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

#define CHECK(call) \
{   \
    const hipError_t error = call; \
    if (error != hipSuccess)    \
    {   \
    printf("Error: %s:%d, ", __FILE__, __LINE__);    \
    printf("code:%d, reason:%s\n", error, hipGetErrorString(error));    \
    exit(1);    \
    }   \
}   \

void initData(int *dataPtr, int size);
double cpuSecond();
void printMatrix(int *mat, const int nx, const int ny);
__global__ void printfThreadIndex(int *mat, const int nx, const int ny);

int main(int argc, char **argv) {
    printf("starting...\n");

    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using device %d: %s\n", dev, deviceProp.name);

    CHECK(hipSetDevice(dev));

    // notice: x,y is more like image coorinate
    // origin is at top left corner
    int nx = 8;
    int ny = 6;
    int nxy = nx*ny;
    int nBytes = nxy * sizeof(float);

    int *h_a;
    h_a = (int*)malloc(nBytes);
    initData(h_a, nxy);
    printMatrix(h_a, nx, ny);

    int *d_a;
    hipMalloc((void**)&d_a, nBytes);
    hipMemcpy(d_a, h_a, nBytes, hipMemcpyHostToDevice);

    dim3 block(4, 2);
    dim3 grid((nx+block.x-1)/block.x, (ny+block.y-1)/block.y);

    printfThreadIndex<<<grid, block>>>(d_a, nx, ny);
    hipDeviceSynchronize();

    hipFree(d_a);
    free(h_a);

    hipDeviceReset();

    return 0;
}

void initData(int *dataPtr, int size) {
    for (int i = 0; i < size; i++) {
        dataPtr[i] = i;
    }
}

double cpuSecond() {
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec*1e-6);
}

void printMatrix(int *mat, const int nx, const int ny) {
    printf("matrix: (%d,%d)\n", nx, ny);
    for (int i = 0; i < nx*ny; i++) {
        printf("%3d", mat[i]);
        if (i%nx == nx-1) {
            printf("\n");
        }
    }
    printf("\n");
}

__global__ void printfThreadIndex(int *mat, const int nx, const int ny) {
    int ix =  threadIdx.x + blockDim.x * blockIdx.x;
    int iy =  threadIdx.y + blockDim.y * blockIdx.y;
    int idx = ix + iy*nx;

    printf("%d %d %d\n", ix, iy, mat[idx]);
}
