#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

#define CHECK(call) \
{   \
    const hipError_t error = call; \
    if (error != hipSuccess)    \
    {   \
    printf("Error: %s:%d, ", __FILE__, __LINE__);    \
    printf("code:%d, reason:%s\n", error, hipGetErrorString(error));    \
    exit(1);    \
    }   \
}   \

void initData(float *dataPtr, int size);
void sumMatrixOnHost(float *pa, float *pb, float *pc, const int size);
void checkResult(float *hRet, float *dRet, const int size);
double cpuSecond();
__global__ void sumMatrixOnGPU(float *pa, float *pb, float *pc, 
                               const int nx, const int ny);

int main(int argc, char **argv) {
    printf("starting...\n");

    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using device %d: %s\n", dev, deviceProp.name);

    CHECK(hipSetDevice(dev));

    int nx = 1 << 12;
    int ny = 1 << 12;
    int nxy = nx*ny;
    size_t nBytes = nxy * sizeof(float);  // 256MB
    printf("vector size: nx=%d ny=%d\n", nx, ny);

    float *h_a, *h_b, *hRet, *dRet;
    h_a = (float *)malloc(nBytes);
    h_b = (float *)malloc(nBytes);
    hRet = (float *)malloc(nBytes);
    dRet = (float *)malloc(nBytes);

    initData(h_a, nxy);
    initData(h_b, nxy);

    float *d_a, *d_b, *d_c;
    hipMalloc((float**)&d_a, nBytes);
    hipMalloc((float**)&d_b, nBytes);
    hipMalloc((float**)&d_c, nBytes);

    // transfer data from host to device
    hipMemcpy(d_a, h_a, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, nBytes, hipMemcpyHostToDevice);

    // invoke kernel at host side
    int dimx = 16;
    int dimy = 16;
    dim3 block(dimx, dimy);
    dim3 grid((nx + block.x -1)/block.x, (ny + block.y - 1)/block.y);

    double start = cpuSecond();
    sumMatrixOnGPU<<<grid, block>>>(d_a, d_b, d_c, nx, ny);
    hipDeviceSynchronize();
    double during = cpuSecond() - start;
    printf("GPU config:grid(%d,%d), block(%d,%d), time elapsed %f\n", 
            grid.x, grid.y, block.x, block.y, during);

    // copy kernel result back to host
    hipMemcpy(dRet, d_c, nBytes, hipMemcpyDeviceToHost);
    
    // host side result
    start = cpuSecond();
    sumMatrixOnHost(h_a, h_b, hRet, nxy);
    during = cpuSecond() - start;
    printf("CPU time elapsed %f\n", during);

    // check result
    checkResult(hRet, dRet, nxy);

    free(h_a);
    free(h_b);
    free(hRet);
    free(dRet);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    
    return 0;
}

void initData(float *dataPtr, int size) {
    time_t t;
    srand((unsigned int)time(&t));
    for (int i = 0; i < size; i++) {
        dataPtr[i] = (float)(rand() & 0xFF) / 10.0f;
    }
}

void sumMatrixOnHost(float *pa, float *pb, float *pc, const int size) {
    for (int i = 0; i < size; i++) {
        pc[i] = pa[i] + pb[i];
    }
}

__global__ void sumMatrixOnGPU(float *pa, float *pb, float *pc,
                               const int nx, const int ny) {
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = ix + iy*nx;
    if (idx < nx*ny)
        pc[idx] = pa[idx] + pb[idx];
}

void checkResult(float *hRet, float *dRet, const int size) {
    double epsilon = 1e-8;
    bool match = 1;
    for (int i = 0; i < size; i++) {
        if (abs(hRet[i]-dRet[i]) > epsilon) {
            match = 0;
            printf("Array not match! idx=%d, host=%5.2f, gpu=%5.2f\n", i, hRet[i], dRet[i]);
            break;
        }
    }

    if (match) {
        printf("Array match!\n");
    }
}

double cpuSecond() {
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec*1e-6);
}



